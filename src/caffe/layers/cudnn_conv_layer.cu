
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>

#include "caffe/filler.hpp"
#include "caffe/layers/cudnn_conv_layer.hpp"
#include "caffe/net.hpp"
#include "caffe/solver.hpp"

namespace caffe {

template<typename Ftype, typename Btype>
void CuDNNConvolutionLayer<Ftype, Btype>::Forward_gpu(const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  const Ftype* weight = this->blobs_[0]->template gpu_data<Ftype>();
  GPUMemory::Workspace& ws = workspace(Caffe::current_device());
  for (int i = 0; i < bottom.size(); ++i) {
    const Ftype* bottom_data = bottom[i]->gpu_data<Ftype>();
    Ftype* top_data = top[i]->mutable_gpu_data<Ftype>();
    // Filters.
    CUDNN_CHECK(cudnnConvolutionForward(Caffe::cudnn_handle(),
        cudnn::dataType<Ftype>::one, fwd_bottom_descs_[i], bottom_data,
        fwd_filter_desc_, weight,
        fwd_conv_descs_[i], fwd_algo_[i], ws.data(), ws.size(),
        cudnn::dataType<Ftype>::zero, fwd_top_descs_[i], top_data));
    if (this->bias_term_) {
      const Ftype* bias_data = this->blobs_[1]->template gpu_data<Ftype>();
      CUDNN_CHECK(cudnnAddTensor(Caffe::cudnn_handle(),
          cudnn::dataType<Ftype>::one,
          fwd_bias_desc_, bias_data,
          cudnn::dataType<Ftype>::one,
          fwd_top_descs_[i], top_data));
    }
    CUDA_CHECK(cudaStreamSynchronize(Caffe::thread_stream()));
  }  // end of for i
  const Solver* psolver = this->parent_solver();
  if (psolver == nullptr || psolver->iterations_sized() > 0) {
    // Possibly use faster algorithms by allowing larger workspace.
    use_modest_workspace_ = false;
  } else {
    Net* pnet = this->parent_net();
    if (pnet == nullptr || pnet->infer_count() > 0) {
      // Same as above in test flow
      use_modest_workspace_ = false;
    }
  }
}

template <typename Ftype, typename Btype>
void CuDNNConvolutionLayer<Ftype, Btype>::Backward_gpu(const vector<Blob*>& top,
    const vector<bool>& propagate_down, const vector<Blob*>& bottom) {
  GPUMemory::Workspace& ws = workspace(Caffe::current_device());

  // compute dE/dB = sum_c(dE/dy)
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    Btype* bias_diff = this->blobs_[1]->template mutable_gpu_diff<Btype>();
    for (int i = 0; i < top.size(); ++i) {
      Btype* top_diff = top[i]->mutable_gpu_diff<Btype>();
      CUDNN_CHECK(cudnnConvolutionBackwardBias(Caffe::cudnn_handle(),
          cudnn::dataType<Btype>::one, bwd_top_descs_[i], top_diff,
          cudnn::dataType<Btype>::one, bwd_bias_desc_, bias_diff));
      CUDA_CHECK(cudaStreamSynchronize(Caffe::thread_stream()));
    }  // end of i
  }  // end of dB

  // compute dE/dW = dY * X
  if (this->param_propagate_down_[0]) {
    Btype* weight_diff = this->blobs_[0]->template mutable_gpu_diff<Btype>();
    for (int i = 0; i < top.size(); ++i) {
      Btype* top_diff = top[i]->mutable_gpu_diff<Btype>();
      const Btype* bottom_data = bottom[i]->gpu_data<Btype>();
      CUDNN_CHECK(cudnnConvolutionBackwardFilter(Caffe::cudnn_handle(),
          cudnn::dataType<Btype>::one, bwd_bottom_descs_[i], bottom_data,
          bwd_top_descs_[i], top_diff,
          bwd_conv_filter_descs_[i], bwd_filter_algo_[i], ws.data(), ws.size(),
          cudnn::dataType<Btype>::one, bwd_filter_desc_, weight_diff));
      CUDA_CHECK(cudaStreamSynchronize(Caffe::thread_stream()));
    }  // end of i
  }

  // Backward propagate grad wrt bottom data dE/dX= dE/dY * W
  const Btype* weight = this->blobs_[0]->template gpu_data<Btype>();
  for (int i = 0; i < top.size(); ++i) {
    if (propagate_down[i]) {
      Btype* top_diff = top[i]->mutable_gpu_diff<Btype>();
      Btype* bottom_diff = bottom[i]->mutable_gpu_diff<Btype>();
      CUDNN_CHECK(cudnnConvolutionBackwardData(Caffe::cudnn_handle(),
          cudnn::dataType<Btype>::one, bwd_filter_desc_, weight,
          bwd_top_descs_[i], top_diff,
          bwd_conv_data_descs_[i],
          bwd_data_algo_[i], ws.data(), ws.size(),
          cudnn::dataType<Btype>::zero, bwd_bottom_descs_[i], bottom_diff));
      CUDA_CHECK(cudaStreamSynchronize(Caffe::thread_stream()));
    }  // end if propagate down
  }  // end for i
}

INSTANTIATE_LAYER_GPU_FUNCS_FB(CuDNNConvolutionLayer);

}  // namespace caffe
#endif
